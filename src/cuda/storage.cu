#include "hip/hip_runtime.h"
#include <storage.cuh>
#include <utils.cuh>

#include <hiprand/hiprand_kernel.h>
#include <>
#include <thrust/reduce.h>

#include <cmath>
#include <exception>

Storage::Storage(thrust::host_vector<std::size_t> shape, float value = 0)
    : shape(shape) {
  std::size_t size = thrust::reduce(shape.begin(), shape.end(), (std::size_t)1,
                                    thrust::multiplies<std::size_t>());
  this->data.resize(size, value);
}

Storage::Storage(std::initializer_list<std::size_t> shape, float value) {
  this->shape = thrust::device_vector<std::size_t>(shape.begin(), shape.end());
  std::size_t size =
      thrust::reduce(this->shape.begin(), this->shape.end(), (std::size_t)1,
                     thrust::multiplies<std::size_t>());
  this->data.resize(size, value);
}

Storage::Storage(thrust::host_vector<std::size_t> shape,
                 thrust::device_vector<float> &&data)
    : shape(shape) {
  std::size_t size =
      thrust::reduce(this->shape.begin(), this->shape.end(), (std::size_t)1,
                     thrust::multiplies<std::size_t>());
  CHECK_EQ(size, data.size(), "Storage: error size");
  this->data = std::move(data);
}

__global__ void storage_xavier(float *a, std::size_t size, float scale) {
  std::size_t index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    hiprandState s;
    hiprand_init(1234, index, 0, &s);
    a[index] = (hiprand_uniform(&s) * 2 - 1) * scale;
  }
}

void Storage::xavier(size_t in_size, size_t out_size) {
  float *a_ptr = thrust::raw_pointer_cast(this->data.data());
  std::size_t size = this->data.size();
  std::size_t block_size = ceil((float)(size) / BLOCK_SIZE);
  float scale = std::sqrt((double)6) / std::sqrt((float)(in_size) + out_size);
  storage_xavier<<<block_size, BLOCK_SIZE>>>(a_ptr, size, scale);

  CUDA_POST_KERNEL_CHECK;
}