#include "hip/hip_runtime.h"
﻿#include <storage.cuh>
#include <utils.cuh>

#include <hiprand/hiprand_kernel.h>
#include <>
#include <thrust/reduce.h>

#include <cmath>
#include <exception>

Storage::Storage() {}

Storage::Storage(const std::vector<int> &_shape, float value)
    : shape(_shape.begin(), _shape.end()) {
  int size = 1;
  for (int i = 0; i < _shape.size(); i++) {
    size *= _shape[i];
  }

  this->data.resize(size, value);
}

Storage::Storage(const std::vector<int> &_shape,
                 const std::vector<float> &_data)
    : shape(_shape.begin(), _shape.end()), data(_data.begin(), _data.end()) {
  this->check_size();
}

Storage::Storage(const std::vector<int> &_shape,
                 thrust::device_vector<float>::const_iterator begin,
                 thrust::device_vector<float>::const_iterator end)
    : shape(_shape.begin(), _shape.end()), data(begin, end) {
  this->check_size();
}

Storage::Storage(const thrust::device_vector<int> &_shape, float value)
    : shape(_shape.begin(), _shape.end()) {
  int size = 1;
  for (int i = 0; i < _shape.size(); i++) {
    size *= _shape[i];
  }

  this->data.resize(size, value);
}

Storage::Storage(const thrust::device_vector<int> &_shape,
                 const thrust::device_vector<float> &_data)
    : shape(_shape.begin(), _shape.end()), data(_data.begin(), _data.end()) {
  this->check_size();
}

Storage::Storage(const thrust::device_vector<int> &_shape,
                 thrust::device_vector<float>::const_iterator begin,
                 thrust::device_vector<float>::const_iterator end)
    : shape(_shape.begin(), _shape.end()), data(begin, end) {
  this->check_size();
}

Storage::Storage(const Storage &other) {
  this->shape.resize(other.shape.size());
  thrust::copy(other.shape.begin(), other.shape.end(), this->shape.begin());

  this->data.resize(other.data.size());
  thrust::copy(other.data.begin(), other.data.end(), this->data.begin());
}

Storage &Storage::operator=(const Storage &other) {
  if (this != &other) {
    this->shape.resize(other.shape.size());
    thrust::copy(other.shape.begin(), other.shape.end(), this->shape.begin());

    this->data.resize(other.data.size());
    thrust::copy(other.data.begin(), other.data.end(), this->data.begin());
  }

  return *this;
}

Storage::Storage(Storage &&other) {
  this->data.swap(other.data);
  this->shape.swap(other.shape);
}

Storage &Storage::operator=(Storage &&other) {
  if (this != &other) {
    this->data.swap(other.data);
    this->shape.swap(other.shape);
  }
  return *this;
}

void Storage::check_size() {
  CHECK_EQ(true, this->shape.size() >= 2, "Storage: error, shape.size() < 2");
  int size = thrust::reduce(this->shape.begin(), this->shape.end(), (int)1,
                            thrust::multiplies<int>());
  CHECK_EQ(size, this->data.size(), "Storage: error size");
}

void Storage::reshape(const std::vector<int> &_shape) {
  this->shape.assign(_shape.begin(), _shape.end());
  this->check_size();
}

void Storage::reshape(const thrust::device_vector<int> &_shape) {
  this->shape.assign(_shape.begin(), _shape.end());
  this->check_size();
}

std::vector<int> Storage::get_shape() {
  return std::vector<int>(this->shape.begin(), this->shape.end());
}

std::vector<float> Storage::get_data() {
  return std::vector<float>(this->data.begin(), this->data.end());
}

__global__ void storage_xavier(float *a, int size, float scale) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  if (index < size) {
    hiprandState s;
    hiprand_init(1234, index, 0, &s);
    a[index] = (hiprand_uniform(&s) * 2 - 1) * scale;
  }
}

void Storage::xavier(size_t in_size, size_t out_size) {
  float *a_ptr = thrust::raw_pointer_cast(this->data.data());
  int size = this->data.size();
  int grid_size = ceil((float)(size) / BLOCK_SIZE);

  float scale = std::sqrt((float)6) / std::sqrt((float)(in_size) + out_size);
  storage_xavier<<<grid_size, BLOCK_SIZE>>>(a_ptr, size, scale);

  CUDA_POST_KERNEL_CHECK;
}