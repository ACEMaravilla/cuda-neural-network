#include "hip/hip_runtime.h"
#include <linear.cuh>
#include <memory>

void operator_linear(const Storage *inputs, const Storage *weights,
                     Storage *output) {
  operator_matmul(inputs, weights, output);
}

void operator_d_linear(const Storage *outputs_grad, const Storage *inputs,
                       const Storage *weights, Storage *weights_grad,
                       Storage *inputs_grad) {
  Storage weights_transpose({weights->get_shape()[1], weights->get_shape()[0]});
  operator_transpose(weights, &weights_transpose);

  Storage inputs_transpose({inputs->get_shape()[1], inputs->get_shape()[0]});
  operator_transpose(inputs, &inputs_transpose);

  // Y = X * W
  // dL/dX = dL/dY * W^T
  // dL/dW = X^T * dL/dY
  operator_matmul(outputs_grad, &weights_transpose, inputs_grad);
  operator_matmul(&inputs_transpose, outputs_grad, weights_grad);
}

__global__ void operator_bias_h(const float *inputs, const float *bias,
                                float *output, int width, int size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < size) {
    int col = index % width;
    output[index] = inputs[index] + bias[col];
  }
}

void operator_linear_bias(const Storage *inputs, const Storage *bias,
                          Storage *output) {
  const float *inputs_ptr = thrust::raw_pointer_cast(inputs->get_data().data());
  const float *bias_ptr = thrust::raw_pointer_cast(bias->get_data().data());
  float *output_ptr = thrust::raw_pointer_cast(output->get_data().data());

  int size = inputs->get_data().size();
  int grid_size = ceil((float)(size) / BLOCK_SIZE);
  int width = bias->get_data().size();
  operator_bias_h<<<grid_size, BLOCK_SIZE>>>(inputs_ptr, bias_ptr, output_ptr,
                                             width, size);

  CUDA_POST_KERNEL_CHECK;
}

void operator_d_linear_bias(const Storage *outputs_grad, Storage *bias_grad) {
  operator_sum(outputs_grad, 0, bias_grad);
}

Linear::Linear(int in_size, int out_size, bool is_bias)
    : in_size(in_size), out_size(out_size), is_bias(is_bias) {
  this->weights.reset(new Storage({in_size, out_size}));
  this->weights_grad.reset(new Storage({in_size, out_size}));
  this->weights->xavier(in_size, out_size);

  if (this->is_bias) {
    this->bias.reset(new Storage({1, out_size}));
    this->bias_grad.reset(new Storage({1, out_size}));
    this->bias->xavier(in_size, out_size);
  }
}

std::vector<std::pair<Storage *, Storage *>> Linear::parameters() {
  if (this->is_bias) {
    return {std::make_pair(this->weights.get(), this->weights_grad.get()),
            std::make_pair(this->bias.get(), this->bias_grad.get())};
  } else {
    return {std::make_pair(this->weights.get(), this->weights_grad.get())};
  }
}

void Linear::forward() {
  const Storage *input = this->pre->get_output();
  std::vector<int> output_shape = {input->get_shape()[0], this->out_size};

  INIT_STORAGE(this->output, output_shape);

  operator_linear(input, this->weights.get(), this->output.get());
  if (this->bias) {
    operator_linear_bias(this->output.get(), this->bias.get(),
                         this->output.get());
  }
}

void Linear::backward() {
  const Storage *input = this->pre->get_output();
  const Storage *output_grad = this->next->get_grad();

  INIT_STORAGE(this->grad, input->get_shape());

  if (this->bias) {
    operator_d_linear_bias(output_grad, this->bias_grad.get());
  }

  operator_d_linear(output_grad, input, this->weights.get(),
                    this->weights_grad.get(), this->grad.get());
}