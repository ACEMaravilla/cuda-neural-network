#include "hip/hip_runtime.h"
#include <linear.cuh>
#include <memory>

Storage *operator_linear(const Storage *inputs, const Storage *weights) {
  return operator_matmul(inputs, weights);
}

Storage *operator_d_linear(const Storage *outputs_grad, const Storage *inputs,
                           const Storage *weights, Storage *weights_grad) {
  std::unique_ptr<Storage> weights_transpose(operator_transpose(weights, 0, 1));
  std::unique_ptr<Storage> inputs_transpose(operator_transpose(inputs, 0, 1));

  // Y = X * W
  // dL/dX = dL/dY * W^T
  // dL/dW = X^T * dL/dY
  Storage *inputs_grad = operator_matmul(outputs_grad, weights_transpose.get());
  std::unique_ptr<Storage> w_grad(
      operator_matmul(inputs_transpose.get(), outputs_grad));
  *weights_grad = std::move(*w_grad.get());

  return inputs_grad;
}

__global__ void operator_bias_h(const float *inputs, const float *bias,
                                float *output, std::size_t width,
                                std::size_t size) {
  std::size_t index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < size) {
    std::size_t col = index % width;
    output[index] = inputs[index] + bias[col];
  }
}

Storage *operator_bias(const Storage *inputs, const Storage *bias) {
  const float *inputs_ptr = thrust::raw_pointer_cast(input1->data.data());
  const float *bias_ptr = thrust::raw_pointer_cast(bias->data.data());
  Storage *output = new Storage(input1->shape);
  float *output_ptr = thrust::raw_pointer_cast(output->data.data());

  std::size_t size = input1->data.size();
  std::size_t grid_size = ceil((float)(size) / BLOCK_SIZE);
  operator_bias_h<<<grid_size, BLOCK_SIZE>>>(inputs_ptr, bias_ptr, output_ptr,
                                             bias->data.size(), size);

  CUDA_POST_KERNEL_CHECK;
  return output;
}

Storage *operator_d_bias(const Storage *outputs_grad, Storage *bias_grad) {
  *bias_grad = *outputs_grad;
}