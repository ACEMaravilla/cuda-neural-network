#include "hip/hip_runtime.h"
#include <linear.cuh>
#include <memory>

Storage *operator_linear(const Storage *inputs, const Storage *weights) {
  return operator_matmul(inputs, weights);
}

Storage *operator_d_linear(const Storage *outputs_grad, const Storage *inputs,
                           const Storage *weights, Storage *weights_grad) {
  std::unique_ptr<Storage> weights_transpose(operator_transpose(weights, 0, 1));
  std::unique_ptr<Storage> inputs_transpose(operator_transpose(inputs, 0, 1));

  // Y = X * W
  // dL/dX = dL/dY * W^T
  // dL/dW = X^T * dL/dY
  Storage *inputs_grad = operator_matmul(outputs_grad, weights_transpose.get());
  std::unique_ptr<Storage> w_grad(
      operator_matmul(inputs_transpose.get(), outputs_grad));
  *weights_grad = std::move(*w_grad.get());

  return inputs_grad;
}

__global__ void operator_bias_h(const float *inputs, const float *bias,
                                float *output, int width, int size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < size) {
    int col = index % width;
    output[index] = inputs[index] + bias[col];
  }
}

Storage *operator_bias(const Storage *inputs, const Storage *bias) {
  const float *inputs_ptr = thrust::raw_pointer_cast(inputs->data.data());
  const float *bias_ptr = thrust::raw_pointer_cast(bias->data.data());
  Storage *output = new Storage(inputs->shape);
  float *output_ptr = thrust::raw_pointer_cast(output->data.data());

  int size = inputs->data.size();
  int grid_size = ceil((float)(size) / BLOCK_SIZE);
  operator_bias_h<<<grid_size, BLOCK_SIZE>>>(inputs_ptr, bias_ptr, output_ptr,
                                             bias->data.size(), size);

  CUDA_POST_KERNEL_CHECK;
  return output;
}

Storage *operator_d_bias(const Storage *outputs_grad, Storage *bias_grad) {
  *bias_grad = *outputs_grad;
  return new Storage(*outputs_grad);
}