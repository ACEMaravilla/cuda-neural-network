#include "hip/hip_runtime.h"
#include <max_pool.cuh>

__global__ void operator_max_pool_h(
    const int nthreads, const float* const bottom_data, const int num,
    const int channels, const int height, const int width,
    const int pooled_height, const int pooled_width, const int kernel_h,
    const int kernel_w, const int stride_h, const int stride_w, const int pad_h,
    const int pad_w, float* const top_data, int* mask) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // output location
    const int pw = index % pooled_width;
    const int ph = (index / pooled_width) % pooled_height;
    const int c = (index / pooled_width / pooled_height) % channels;
    const int n = index / pooled_width / pooled_height / channels;

    // pooled range
    int hstart = ph * stride_h - pad_h;
    int wstart = pw * stride_w - pad_w;
    const int hend = fminf(hstart + kernel_h, height);
    const int wend = fminf(wstart + kernel_w, width);
    hstart = fmaxf(hstart, 0);
    wstart = fmaxf(wstart, 0);

    // get max value postion
    float maxval = -FLT_MAX;
    int maxidx = -1;
    const float* const bottom_slice =
        bottom_data + (n * channels + c) * height * width;
    for (int h = hstart; h < hend; ++h) {
      for (int w = wstart; w < wend; ++w) {
        if (bottom_slice[h * width + w] > maxval) {
          maxidx = h * width + w;
          maxval = bottom_slice[maxidx];
        }
      }
    }
    // output
    top_data[index] = maxval;

    // record idx
    mask[index] = maxidx;
  }
}

__global__ void operator_d_max_pool_h(
    const int nthreads, const float* const top_diff, const int* const mask,
    const int num, const int channels, const int height, const int width,
    const int pooled_height, const int pooled_width, const int kernel_h,
    const int kernel_w, const int stride_h, const int stride_w, const int pad_h,
    const int pad_w, float* const bottom_diff) {
  CUDA_KERNEL_LOOP(index, nthreads) {
    // find out the local index
    // find out the local offset
    const int w = index % width;
    const int h = (index / width) % height;
    const int c = (index / width / height) % channels;
    const int n = index / width / height / channels;

    // pooled range
    const int phstart =
        (h + pad_h < kernel_h) ? 0 : (h + pad_h - kernel_h) / stride_h + 1;
    const int phend = fminf((h + pad_h) / stride_h + 1, pooled_height);
    const int pwstart =
        (w + pad_w < kernel_w) ? 0 : (w + pad_w - kernel_w) / stride_w + 1;
    const int pwend = fminf((w + pad_w) / stride_w + 1, pooled_width);
    float gradient = 0;
    const int offset = (n * channels + c) * pooled_height * pooled_width;
    const float* const top_diff_slice = top_diff + offset;

    // get max value idx
    const int* const mask_slice = mask + offset;
    for (int ph = phstart; ph < phend; ++ph) {
      for (int pw = pwstart; pw < pwend; ++pw) {
        if (mask_slice[ph * pooled_width + pw] == h * width + w) {
          gradient += top_diff_slice[ph * pooled_width + pw];
        }
      }
    }

    bottom_diff[index] = gradient;
  }
}
