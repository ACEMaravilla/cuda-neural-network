#include "hip/hip_runtime.h"
#include <softmax.cuh>

__global__ void operator_log_softmax_h(const float *input1, float *output,
                                       const int *input1_shape, int input1_dims,
                                       const int *temp_shape, int dim,
                                       int dim_stride, int size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < size) {
    int length = input1_shape[dim];

    int *loc = new int[input1_dims];
    index2loc(index, temp_shape, input1_dims - 1, loc);
    for (int i = input1_dims - 1; i > dim; i--) {
      loc[i] = loc[i - 1];
    }
    loc[dim] = 0;
    int base = loc2index(loc, input1_shape, input1_dims);
    delete[] loc;

    float max_ = -FLT_MAX;
    for (int i = 0; i < length; ++i) {
      max_ = fmaxf(max_, input1[base + i * dim_stride]);
    }

    double logsum = 0;
    for (int i = 0; i < length; ++i) {
      logsum += expf(input1[base + i * dim_stride] - max_);
    }
    logsum = max_ + logf(logsum);

    for (int i = 0; i < length; ++i) {
      output[base + i * dim_stride] = input1[base + i * dim_stride] - logsum;
    }
  }
}

void operator_log_softmax(const Storage *input1, int dim, Storage *outputs) {
  const float *input1_ptr = thrust::raw_pointer_cast(input1->data.data());
  const int *input1_shape_ptr = thrust::raw_pointer_cast(input1->shape.data());
  float *output_ptr = thrust::raw_pointer_cast(outputs->data.data());

  outputs->data.resize(input1->data.size());
  outputs->reshape(input1->shape);

  thrust::device_vector<int> temp_shape(input1->shape);
  temp_shape.erase(temp_shape.begin() + dim);
  int *temp_shape_ptr = thrust::raw_pointer_cast(temp_shape.data());

  int input1_dims = input1->shape.size();
  int dim_stride = 1;
  for (int i = dim + 1; i < input1_dims; i++) {
    dim_stride *= input1_shape_ptr[i];
  }

  int size = input1->data.size() / input1->shape[dim];
  int grid_size = ceil((float)(size) / BLOCK_SIZE);
  operator_log_softmax_h<<<grid_size, BLOCK_SIZE>>>(
      input1_ptr, output_ptr, input1_shape_ptr, input1_dims, temp_shape_ptr,
      dim, dim_stride, size);

  CUDA_POST_KERNEL_CHECK;
}

__global__ void operator_d_log_softmax_h(const float *output_grads,
                                         const float *input1,
                                         const int *input1_shape,
                                         const int *temp_shape, int input1_dims,
                                         int dim, int dim_stride, int size,
                                         float *input1_grads) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < size) {
    int length = input1_shape[dim];

    int *loc = new int[input1_dims];
    index2loc(index, temp_shape, input1_dims - 1, loc);
    for (int i = input1_dims - 1; i > dim; i--) {
      loc[i] = loc[i - 1];
    }
    loc[dim] = 0;
    int base = loc2index(loc, input1_shape, input1_dims);
    delete[] loc;

    float max_ = -FLT_MAX;
    for (int i = 0; i < length; ++i) {
      max_ = fmaxf(max_, input1[base + i * dim_stride]);
    }

    double logsum = 0;
    for (int i = 0; i < length; ++i) {
      logsum += expf(input1[base + i * dim_stride] - max_);
    }
    logsum = max_ + logf(logsum);

    // sum(dL/dY) = dL/dY * 1_n
    double dldysum = 0;
    for (int i = 0; i < length; ++i) {
      dldysum += output_grads[base + i * dim_stride];
    }

    // dL/dY - sum(dL/dY) * exp(x) / sum(exp(x))
    for (int i = 0; i < length; ++i) {
      float x = input1[base + i * dim_stride];
      input1_grads[base + i * dim_stride] =
          output_grads[base + i * dim_stride] - dldysum * expf(x - logsum);
    }
  }
}

// Y = log_softmax(X) = x - log(exp(X) * 1_n) * 1_n^T
// dL/dX = dL/dY - (dL/dY * 1_n * exp(x)) / (exp(x) * 1_n)
void operator_d_log_softmax(const Storage *output_grads, const Storage *input1,
                            int dim, Storage *inputs_grad) {
  const float *input1_ptr = thrust::raw_pointer_cast(input1->data.data());
  const int *input1_shape_ptr = thrust::raw_pointer_cast(input1->shape.data());
  const float *output_grads_ptr =
      thrust::raw_pointer_cast(output_grads->data.data());

  float *input1_grads_ptr = thrust::raw_pointer_cast(inputs_grad->data.data());
  inputs_grad->data.resize(input1->data.size());
  inputs_grad->reshape(input1->shape);

  thrust::device_vector<int> temp_shape(input1->shape);
  temp_shape.erase(temp_shape.begin() + dim);
  int *temp_shape_ptr = thrust::raw_pointer_cast(temp_shape.data());

  int input1_dims = input1->shape.size();
  int dim_stride = 1;
  for (int i = dim + 1; i < input1_dims; i++) {
    dim_stride *= input1_shape_ptr[i];
  }

  int size = input1->data.size() / input1->shape[dim];
  int grid_size = ceil((float)(size) / BLOCK_SIZE);
  operator_d_log_softmax_h<<<grid_size, BLOCK_SIZE>>>(
      output_grads_ptr, input1_ptr, input1_shape_ptr, temp_shape_ptr,
      input1_dims, dim, dim_stride, size, input1_grads_ptr);

  CUDA_POST_KERNEL_CHECK;
}