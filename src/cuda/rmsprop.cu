#include "hip/hip_runtime.h"
#include <rmsprop.cuh>

#include <thrust/device_vector.h>
#include <thrust/transform.h>

#include <memory>

struct rms_suqare_grads_functor {
  const float b;

  rms_suqare_grads_functor(float _b) : b(_b) {}

  __host__ __device__ float operator()(const float &x, const float &y) const {
    float sq = powf(y, 2);
    return b * x + (1 - b) * sq;
  }
};

struct rms_grads_functor {
  const float a;

  rms_grads_functor(float _a) : a(_a) {}

  __host__ __device__ float operator()(const float &x, const float &y) const {
    return a * x / (sqrtf(y) + 1e-10);
  }
};

struct l2_grads_functor {
  const float l2;

  l2_grads_functor(float _l2) : l2(_l2) {}

  __host__ __device__ float operator()(const float &x, const float &y) const {
    return x + l2 * y;
  }
};

void rmsprop_update(Storage *square_grads, Storage *weights,
                    const Storage *grads, float learning_rate, float l2,
                    float beta) {
  // reduce grads
  Storage *reduce_grads = nullptr;
  if (grads->data.size() > weights->data.size()) {
    reduce_grads = new Storage();
    operator_sum(grads, 0, reduce_grads);
    grads = reduce_grads;
  }

  CHECK_EQ(square_grads->data.size(), grads->data.size(),
           "RMSProp: grads size error 1");
  CHECK_EQ(weights->data.size(), grads->data.size(),
           "RMSProp: grads size error 2");

  // add L2 weights grads
  l2_grads_functor l2f(l2);
  thrust::device_vector<float> l2_grads(grads->data.size());
  thrust::transform(grads->data.begin(), grads->data.end(),
                    weights->data.begin(), l2_grads.begin(), l2f);

  // rms grads
  rms_suqare_grads_functor sgf(beta);
  thrust::transform(square_grads->data.begin(), square_grads->data.end(),
                    l2_grads.begin(), square_grads->data.begin(), sgf);

  rms_grads_functor gf(learning_rate);
  thrust::device_vector<float> rms_grads(l2_grads.size());
  thrust::transform(l2_grads.begin(), l2_grads.end(),
                    square_grads->data.begin(), rms_grads.begin(), gf);

  // update weights
  thrust::transform(weights->data.begin(), weights->data.end(),
                    rms_grads.begin(), weights->data.begin(),
                    thrust::minus<float>());

  // clean
  if (reduce_grads != nullptr) {
    delete reduce_grads;
  }
}
