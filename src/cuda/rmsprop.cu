#include "hip/hip_runtime.h"
﻿#include <rmsprop.cuh>

#include <thrust/device_vector.h>
#include <thrust/transform.h>

#include <memory>

struct rms_suqare_grads_functor {
  const float b;

  rms_suqare_grads_functor(float _b) : b(_b) {}

  __host__ __device__ float operator()(const float &x, const float &y) const {
    float sq = powf(y, 2);
    return b * x + (1 - b) * sq;
  }
};

struct rms_grads_functor {
  const float a;

  rms_grads_functor(float _a) : a(_a) {}

  __host__ __device__ float operator()(const float &x, const float &y) const {
    return a * x / (sqrtf(y) + 1e-10);
  }
};

struct l2_grads_functor {
  const float l2;

  l2_grads_functor(float _l2) : l2(_l2) {}

  __host__ __device__ float operator()(const float &x, const float &y) const {
    return x + l2 * 2 * y;
  }
};

void rmsprop_update(Storage *square_grads, Storage *weights,
                    const Storage *grads, float learning_rate, float l2,
                    float beta) {
  // need reduce
  const Storage *reduce_grads = grads;
  if (grads->data.size() > weights->data.size()) {
    reduce_grads = operator_sum(grads, 0);
  }

  CHECK_EQ(square_grads->data.size(), reduce_grads->data.size(),
           "RMSProp: grads size error 1");
  CHECK_EQ(weights->data.size(), reduce_grads->data.size(),
           "RMSProp: grads size error 2");

  // add L2 weights grads
  l2_grads_functor l2f(l2);
  thrust::device_vector<float> l2_grads(reduce_grads->data.size());
  thrust::transform(reduce_grads->data.begin(), reduce_grads->data.end(),
                    weights->data.begin(), l2_grads.begin(), l2f);

  // rms grads
  rms_suqare_grads_functor sgf(beta);
  thrust::device_vector<float> new_square_grads(reduce_grads->data.size());
  thrust::transform(square_grads->data.begin(), square_grads->data.end(),
                    l2_grads.begin(), new_square_grads.begin(), sgf);
  square_grads->data = std::move(new_square_grads);

  rms_grads_functor gf(learning_rate);
  thrust::device_vector<float> rms_grads(reduce_grads->data.size());
  thrust::transform(reduce_grads->data.begin(), reduce_grads->data.end(),
                    square_grads->data.begin(), rms_grads.begin(), gf);

  // update weights
  thrust::device_vector<float> new_weights(weights->data.size());
  thrust::transform(weights->data.begin(), weights->data.end(),
                    rms_grads.begin(), new_weights.begin(),
                    thrust::minus<float>());
  weights->data = std::move(new_weights);

  // clean
  if (grads->data.size() > weights->data.size()) {
    delete reduce_grads;
  }
}
