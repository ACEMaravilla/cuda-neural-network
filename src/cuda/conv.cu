#include "hip/hip_runtime.h"
#include <conv.cuh>

#include <thrust/copy.h>
#include <thrust/device_vector.h>

#include <cstdlib>
#include <memory>
#include <vector>

// inputs: N*C*H*W
// filters: C_out*C_in*K_h*K_w
void operator_conv(const Storage *inputs, const Storage *filters, Storage *cols,
                   const int pad_h, const int pad_w, const int stride_h,
                   const int stride_w, Storage *output) {
  CHECK_EQ(inputs->get_shape().size(), 4, "operator_conv: inputs shape error");
  CHECK_EQ(filters->get_shape().size(), 4, "operator_conv: filters shape error");

  int width = *(inputs->get_shape().rbegin());
  int height = *(inputs->get_shape().rbegin() + 1);
  int channel_in = *(inputs->get_shape().rbegin() + 2);
  int batch_size = *(inputs->get_shape().rbegin() + 3);

  int kernel_w = *(filters->get_shape().rbegin());
  int kernel_h = *(filters->get_shape().rbegin() + 1);
  int channel_out = *(filters->get_shape().rbegin() + 3);

  CHECK_EQ(*(filters->get_shape().rbegin() + 2), channel_in,
           "operator_conv: channel size error");

  int height_col = (height + 2 * pad_h - kernel_h) / stride_h + 1;
  int width_col = (width + 2 * pad_w - kernel_w) / stride_w + 1;

  int batch_im_stride = channel_in * height * width;
  int batch_col_stride =
      channel_in * kernel_h * kernel_w * height_col * width_col;

  // im2col
  // [batch_size*(C_in*k_h*k_w)*(height_col * width_col)]
  cols->get_data().resize(batch_size * channel_in * kernel_h * kernel_w * height_col *
                    width_col);
  cols->reshape(
      {batch_size, channel_in * kernel_h * kernel_w, height_col * width_col});

  const float *inputs_ptr = thrust::raw_pointer_cast(inputs->get_data().data());
  const float *filters_ptr = thrust::raw_pointer_cast(filters->get_data().data());
  float *cols_ptr = thrust::raw_pointer_cast(cols->get_data().data());
  for (int i = 0; i < batch_size; i++) {
    im2col(inputs_ptr + i * batch_im_stride, channel_in, height, width,
           kernel_h, kernel_w, pad_h, pad_w, stride_h, stride_w,
           cols_ptr + i * batch_col_stride);
  }

  // matmul
  // Y = F * col
  // [C_out*(C_in*k_h*k_w)] * [(C_in*k_h*k_w)*(height_col*width_col)]
  Storage temp_filters(*filters);
  temp_filters.reshape(
      std::vector<int>{channel_out, channel_in * kernel_h * kernel_w});

  // [batch_size * channel_out * (height_col * width_col)]
  output->get_data().resize(batch_size * channel_out * height_col * width_col);
  output->reshape({batch_size, channel_out, height_col, width_col});
  int batch_output_stride = channel_out * height_col * width_col;

  for (int i = 0; i < batch_size; ++i) {
    auto cols_iter = cols->get_data().begin() + i * batch_col_stride;
    Storage col(std::vector<int>{channel_in * kernel_h * kernel_w,
                                 height_col * width_col},
                cols_iter, cols_iter + batch_col_stride);

    Storage y_temp;
    operator_matmul(&temp_filters, &col, &y_temp);

    auto outputs_iter = output->get_data().begin() + i * batch_output_stride;
    assert(y_temp.get_data().size() == batch_output_stride);
    thrust::copy(y_temp.get_data().begin(), y_temp.get_data().end(), outputs_iter);
  }
}

// Y = F * col
// dL/d_col = F^T * dL/dY
// dL/d_im = col2im(dL/d_col)
// dL/dF = dL/dY * col^T
void operator_d_conv(const Storage *outputs_grad, const Storage *inputs,
                     const Storage *cols, const Storage *filters,
                     const int pad_h, const int pad_w, const int stride_h,
                     const int stride_w, Storage *filters_grad,
                     Storage *inputs_grad) {
  CHECK_EQ(outputs_grad->get_shape().size(), 4,
           "operator_conv: outputs_grad shape error");
  CHECK_EQ(inputs->get_shape().size(), 4, "operator_conv: inputs shape error");
  CHECK_EQ(cols->get_shape().size(), 3, "operator_conv: cols shape error");
  CHECK_EQ(filters->get_shape().size(), 4, "operator_conv: filters shape error");

  Storage *inputs_grad = new Storage(inputs->get_shape());

  int width = *(inputs->get_shape().rbegin());
  int height = *(inputs->get_shape().rbegin() + 1);
  int channel_in = *(inputs->get_shape().rbegin() + 2);
  int batch_size = *(inputs->get_shape().rbegin() + 3);

  int kernel_w = *(filters->get_shape().rbegin());
  int kernel_h = *(filters->get_shape().rbegin() + 1);
  int channel_out = *(filters->get_shape().rbegin() + 3);

  int height_col = (height + 2 * pad_h - kernel_h) / stride_h + 1;
  int width_col = (width + 2 * pad_w - kernel_w) / stride_w + 1;

  // F^T
  Storage filters_temp(*filters);
  filters_temp.reshape(
      std::vector<int>{channel_out, channel_in * kernel_h * kernel_w});
  Storage filters_trans;
  operator_transpose(&filters_temp, 0, 1, &filters_trans);

  // filters grad
  filters_grad->get_data().resize(batch_size * channel_out * channel_in * kernel_h *
                            kernel_w);
  filters_grad->reshape(
      {batch_size, channel_out, channel_in, kernel_h, kernel_w});

  // inputs grad
  inputs_grad->get_data().resize(inputs->get_data().size());
  inputs_grad->reshape(inputs->get_shape());

  // stride
  // int batch_im_stride = channel_in * height * width;
  int batch_col_stride =
      channel_in * kernel_h * kernel_w * height_col * width_col;

  int batch_inputs_grad_stride = channel_in * height * width;
  int batch_filters_grad_stride =
      channel_out * channel_in * kernel_h * kernel_w;
  int batch_outputs_grad_stride = channel_out * height_col * width_col;

  for (int i = 0; i < batch_size; ++i) {
    Storage dl_dy(
        std::vector<int>{channel_out, height_col * width_col},
        outputs_grad->get_data().begin() + i * batch_outputs_grad_stride,
        outputs_grad->get_data().begin() + (i + 1) * batch_outputs_grad_stride);
    // dL/d_col = F^T * dL/dY
    Storage dl_dcol;
    operator_matmul(&filters_trans, &dl_dy, &dl_dcol);

    // dL/d_im = col2im(dL/d_col)
    Storage dl_dim({channel_in, height, width});
    const float *dl_dcol_ptr = thrust::raw_pointer_cast(dl_dcol.get_data().data());
    float *dl_dim_ptr = thrust::raw_pointer_cast(dl_dim.get_data().data());
    col2im(dl_dcol_ptr, channel_in, height, width, kernel_h, kernel_w, pad_h,
           pad_w, stride_h, stride_w, dl_dim_ptr);
    assert(dl_dim.get_data().size() == batch_inputs_grad_stride);
    thrust::copy(dl_dim.get_data().begin(), dl_dim.get_data().end(),
                 inputs_grad->get_data().begin() + i * batch_inputs_grad_stride);

    // dL/dF = dL/dY * col^T
    Storage col(std::vector<int>{channel_in * kernel_h * kernel_w,
                                 height_col * width_col},
                cols->get_data().begin() + i * batch_col_stride,
                cols->get_data().begin() + (i + 1) * batch_col_stride);
    Storage col_t;
    operator_transpose(&col, 0, 1, &col_t);

    Storage dl_df;
    operator_matmul(&dl_dy, &col_t, &dl_df);
    assert(dl_df.get_data().size() == batch_filters_grad_stride);
    thrust::copy(dl_df.get_data().begin(), dl_df.get_data().end(),
                 filters_grad->get_data().begin() + i * batch_filters_grad_stride);
  }
}

__global__ void operator_conv_bias_h(const float *inputs, const float *bias,
                                     float *output, int channel_size,
                                     int channel_stride, int size) {
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < size) {
    int col = (index / channel_stride) % channel_size;
    output[index] = inputs[index] + bias[col];
  }
}

void operator_conv_bias(const Storage *inputs, const Storage *bias,
                        Storage *output) {
  CHECK_EQ(bias->get_data().size(), *(inputs->get_shape().begin() + 1),
           "operator_conv_bias: size error");

  const float *inputs_ptr = thrust::raw_pointer_cast(inputs->get_data().data());
  const float *bias_ptr = thrust::raw_pointer_cast(bias->get_data().data());
  float *output_ptr = thrust::raw_pointer_cast(output->get_data().data());

  int channel_stride =
      *(inputs->get_shape().rbegin()) * *(inputs->get_shape().rbegin() + 1);

  int size = inputs->get_data().size();
  int grid_size = ceil((float)(size) / BLOCK_SIZE);
  operator_conv_bias_h<<<grid_size, BLOCK_SIZE>>>(inputs_ptr, bias_ptr,
                                                  output_ptr, bias->get_data().size(),
                                                  channel_stride, size);

  CUDA_POST_KERNEL_CHECK;
}

void operator_d_conv_bias(const Storage *outputs_grad, Storage *bias_grad,
                          Storage *inputs_grad) {
  // N*C*H*W ==> N*C
  Storage sum3;
  operator_sum(outputs_grad, 3, &sum3);
  Storage sum2;
  operator_sum(&sum3, 2, &sum2);

  *bias_grad = std::move(sum2);
  *inputs_grad = *outputs_grad;
}

// C*H*W >> (C_out*k_h*k_w) * (height_col * width_col)
__global__ void im2col_h(const int n, const float *data_im, const int height,
                         const int width, const int kernel_h,
                         const int kernel_w, const int pad_h, const int pad_w,
                         const int stride_h, const int stride_w,
                         const int height_col, const int width_col,
                         float *data_col) {
  CUDA_KERNEL_LOOP(index, n) {
    const int h_index = index / width_col;
    const int h_col = h_index % height_col;
    const int w_col = index % width_col;
    const int c_im = h_index / height_col;
    const int c_col = c_im * kernel_h * kernel_w;
    const int h_offset = h_col * stride_h - pad_h;
    const int w_offset = w_col * stride_w - pad_w;

    // channel offset
    float *data_col_ptr = data_col;
    data_col_ptr += (c_col * height_col + h_col) * width_col + w_col;
    const float *data_im_ptr = data_im;
    data_im_ptr += (c_im * height + h_offset) * width + w_offset;

    // copy to col
    for (int i = 0; i < kernel_h; ++i) {
      for (int j = 0; j < kernel_w; ++j) {
        int h_im = h_offset + i;
        int w_im = w_offset + j;
        *data_col_ptr =
            (h_im >= 0 && w_im >= 0 && h_im < height && w_im < width)
                ? data_im_ptr[i * width + j]
                : 0;
        data_col_ptr += height_col * width_col;
      }
    }
  }
}

void im2col(const float *data_im, const int channels, const int height,
            const int width, const int kernel_h, const int kernel_w,
            const int pad_h, const int pad_w, const int stride_h,
            const int stride_w, float *data_col) {
  // We are going to launch channels * height_col * width_col kernels, each
  // kernel responsible for copying a single-channel grid.
  int height_col = (height + 2 * pad_h - kernel_h) / stride_h + 1;
  int width_col = (width + 2 * pad_w - kernel_w) / stride_w + 1;
  int num_kernels = channels * height_col * width_col;
  int grid_size = ceil((float)num_kernels / BLOCK_SIZE);

  im2col_h<<<grid_size, BLOCK_SIZE>>>(
      num_kernels, data_im, height, width, kernel_h, kernel_w, pad_h, pad_w,
      stride_h, stride_w, height_col, width_col, data_col);
  CUDA_POST_KERNEL_CHECK;
}

// (C_out*k_h*k_w) * (height_col * width_col) >> C*H*W
__global__ void col2im_h(const int n, const float *data_col, const int height,
                         const int width, const int channels,
                         const int kernel_h, const int kernel_w,
                         const int pad_h, const int pad_w, const int stride_h,
                         const int stride_w, const int height_col,
                         const int width_col, float *data_im) {
  CUDA_KERNEL_LOOP(index, n) {
    float val = 0;
    const int w_im = index % width + pad_w;
    const int h_im = (index / width) % height + pad_h;
    const int c_im = index / (width * height);

    // compute the start and end of the col
    const int w_col_start =
        (w_im < kernel_w) ? 0 : (w_im - kernel_w) / stride_w + 1;
    const int w_col_end = fminf(w_im / stride_w + 1, width_col);
    const int h_col_start =
        (h_im < kernel_h) ? 0 : (h_im - kernel_h) / stride_h + 1;
    const int h_col_end = fminf(h_im / stride_h + 1, height_col);

    // copy to im
    for (int h_col = h_col_start; h_col < h_col_end; h_col += 1) {
      for (int w_col = w_col_start; w_col < w_col_end; w_col += 1) {
        int h_k = (h_im - h_col * stride_h);
        int w_k = (w_im - w_col * stride_w);
        int data_col_index =
            (((c_im * kernel_h + h_k) * kernel_w + w_k) * height_col + h_col) *
                width_col +
            w_col;
        val += data_col[data_col_index];
      }
    }
    data_im[index] = val;
  }
}

void col2im(const float *data_col, const int channels, const int height,
            const int width, const int kernel_h, const int kernel_w,
            const int pad_h, const int pad_w, const int stride_h,
            const int stride_w, float *data_im) {
  int height_col = height + 2 * pad_h - kernel_h / stride_h + 1;
  int width_col = width + 2 * pad_w - kernel_w / stride_w + 1;
  int num_kernels = channels * height * width;
  // To avoid involving atomic operations, we will launch one kernel per
  // bottom dimension, and then in the kernel add up the top dimensions.
  int grid_size = ceil((float)num_kernels / BLOCK_SIZE);
  col2im_h<<<grid_size, BLOCK_SIZE>>>(
      num_kernels, data_col, height, width, channels, kernel_h, kernel_w, pad_h,
      pad_w, stride_h, stride_w, height_col, width_col, data_im);
  CUDA_POST_KERNEL_CHECK;
}
